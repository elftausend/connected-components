
#include <hip/hip_runtime.h>

extern "C" {

    __global__ void labelWithSharedLinksInterleaved(unsigned int* labels, ushort4* links, uchar4* pixels, int width, int height) {
        unsigned int c = blockIdx.x * blockDim.x + threadIdx.x;
        unsigned int r = blockIdx.y * blockDim.y + threadIdx.y;
        if (c >= width || r >= height) {
            return;
        }
        int threshold = 20;

        unsigned int idx = r * width + c;

        uchar4 currentPixel = pixels[idx];
        // if (currentPixel.x == 255 && currentPixel.y == 255 && currentPixel.z == 255) {
        //     return;
        // }

        __shared__ ushort4 sharedLinks[32][33];
        ushort4 currentLink = make_ushort4(0, 0, 0, 0);

        unsigned int rightMove = c < width - 1 ? 1 : 0;
        unsigned int downMove = r < height - 1 ? 1 : 0;
        unsigned int leftMove = c > 0 ? 1 : 0;
        unsigned int upMove = r > 0 ? 1 : 0;

        uchar4 rightPixel = pixels[idx + rightMove];
        uchar4 downPixel = pixels[idx + downMove * width];
        uchar4 leftPixel = pixels[idx - leftMove];
        uchar4 upPixel = pixels[idx - upMove * width];

        int rightPixelDifferenceSum = abs(rightPixel.x - currentPixel.x) + abs(rightPixel.y - currentPixel.y) + abs(rightPixel.z - currentPixel.z);  
        if (rightPixelDifferenceSum < threshold && rightMove) {    
            currentLink.x = 1;
        }

        int downPixelDifferenceSum = abs(downPixel.x - currentPixel.x) + abs(downPixel.y - currentPixel.y) + abs(downPixel.z - currentPixel.z);
        if (downPixelDifferenceSum < threshold && downMove) {    
            currentLink.y = 1;
        }

        int leftPixelDifferenceSum = abs(leftPixel.x - currentPixel.x) + abs(leftPixel.y - currentPixel.y) + abs(leftPixel.z - currentPixel.z);
        if (leftPixelDifferenceSum < threshold && leftMove) {    
            currentLink.z = 1;
        }

        int upPixelDifferenceSum = abs(upPixel.x - currentPixel.x) + abs(upPixel.y - currentPixel.y) + abs(upPixel.z - currentPixel.z);
        if (upPixelDifferenceSum < threshold && upMove) {    
            currentLink.w = 1;
        }

        sharedLinks[threadIdx.y][threadIdx.x] = currentLink;
        __syncthreads();

        for (int i=0; i<5; i++) {
            if (threadIdx.x + currentLink.x < 32) {
                // right
                currentLink.x += sharedLinks[threadIdx.y][threadIdx.x + currentLink.x].x;
            }

            if (threadIdx.y + currentLink.y < 32) {
                // down
                currentLink.y += sharedLinks[threadIdx.y + currentLink.y][threadIdx.x].y;
            }

            if ((int)threadIdx.x - (int)currentLink.z >= 0) {
                // left
                currentLink.z += sharedLinks[threadIdx.y][threadIdx.x - currentLink.z].z;
            }

            if ((int)threadIdx.y - (int)currentLink.w >= 0) {
                // up
                currentLink.w += sharedLinks[threadIdx.y - currentLink.w][threadIdx.x].w;
            }

            sharedLinks[threadIdx.y][threadIdx.x] = currentLink;
            __syncthreads();
        }

        links[idx] = sharedLinks[threadIdx.y][threadIdx.x];

        
        // links[labelIdx] = currentLink;

        labels[idx] = idx;
    }

    __global__ void classifyRootCandidatesShifting(unsigned int* input, ushort4* links, int width, int height) {
        unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
        unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x >= width || y >= height) {
            return;
        }

        int outIdx = y * width + x;
        
        unsigned int currentLabel = input[outIdx];
        ushort4 currentLink = links[outIdx];

        if (currentLink.x == 0 && currentLink.y == 0) {
            unsigned int rootCandidateLabel = (1 << 31) | currentLabel;
            input[outIdx] = rootCandidateLabel;
            return;
        }
    }

    struct PossibleDirections {
        char right;
        char down;
        char left;
        char up;
    };

    __device__ PossibleDirections possibleDirections(char lastDir, ushort4 borderLinks, unsigned int borderLinkIdx, unsigned int* hasVisited, int width, int height) {
            char rightIsPossible = borderLinks.x >= 1 && (borderLinks.y == 0 || borderLinks.w == 0) && lastDir != 2;
            char downIsPossible = borderLinks.y >= 1 && (borderLinks.x == 0 || borderLinks.z == 0) && lastDir != 3;
            char leftIsPossible = borderLinks.z >= 1 && (borderLinks.y == 0 || borderLinks.w == 0) && lastDir != 0;
            char upIsPossible = borderLinks.w >= 1 && (borderLinks.x == 0 || borderLinks.z == 0) && lastDir != 1;
            char isMovePossible = rightIsPossible || downIsPossible || leftIsPossible || upIsPossible;

            // unsigned int nextBorderLinkIdxRight = borderLinkIdx + 1;
            // unsigned int nextBorderLinkIdxDown = borderLinkIdx + width;
            // unsigned int nextBorderLinkIdxLeft = borderLinkIdx - 1;
            // unsigned int nextBorderLinkIdxUp = borderLinkIdx - width;

            return PossibleDirections{rightIsPossible, downIsPossible, leftIsPossible, upIsPossible};

    }

    // use mask and shifting for hasVisited
    __global__ void createBorderPath(unsigned int* labels, ushort4* links, unsigned char* hasVisited, int width, int height) {
        unsigned int c = blockIdx.x * blockDim.x + threadIdx.x;
        unsigned int r = blockIdx.y * blockDim.y + threadIdx.y;

        if (c >= width || r >= height) {
            return;
        }

        int outIdx = r * width + c;
        unsigned int currentLabel = labels[outIdx];

        if (!(currentLabel >> 31)) {
            return;
        }
        unsigned int mask = 0b01111111111111111111111111111111;
        unsigned int rootCandidateLabel = currentLabel & mask;
        unsigned int firstBorderLinkIdx = outIdx;
        unsigned int borderLinkIdx = firstBorderLinkIdx;

        ushort4 borderLinks = links[borderLinkIdx];

        // labels[borderLinkIdx] = 233550;

        char done = false; 
        char lastDir = 1;
        char state = 0;

        int iters = 0;
        while (!done) {

            char rightIsPossible;
            char downIsPossible;
            char leftIsPossible;
            char upIsPossible;

            if (state == 0) {
                rightIsPossible = borderLinks.x >= 1 && (borderLinks.y == 0 || borderLinks.w == 0) && lastDir != 2;
                downIsPossible = borderLinks.y >= 1 && (borderLinks.x == 0 || borderLinks.z == 0) && lastDir != 3;
                leftIsPossible = borderLinks.z >= 1 && (borderLinks.y == 0 || borderLinks.w == 0) && lastDir != 0;
                upIsPossible = borderLinks.w >= 1 && (borderLinks.x == 0 || borderLinks.z == 0) && lastDir != 1;

                char isMovePossible = rightIsPossible || downIsPossible || leftIsPossible || upIsPossible;
                if (!isMovePossible) {
                    ushort4 nextLinksLeft = links[borderLinkIdx - 1];
                    leftIsPossible = nextLinksLeft.x >= 1 && (nextLinksLeft.y == 0 || nextLinksLeft.w == 0) && (lastDir == 3 || lastDir == 1);

                    ushort4 nextLinksRight = links[borderLinkIdx + 1];
                    rightIsPossible = nextLinksRight.z >= 1 && (nextLinksRight.y == 0 || nextLinksRight.w == 0) && (lastDir == 3 || lastDir == 1);
                    
                    ushort4 nextLinksDown = links[borderLinkIdx + width];
                    downIsPossible = nextLinksDown.w >= 1 && (nextLinksDown.x == 0 || nextLinksDown.z == 0) && (lastDir == 0 || lastDir == 2);
                    
                    ushort4 nextLinksUp = links[borderLinkIdx - width];
                    upIsPossible = nextLinksUp.y >= 1 && (nextLinksUp.x == 0 || nextLinksUp.z == 0) && (lastDir == 0 || lastDir == 2);

                    isMovePossible = rightIsPossible || downIsPossible || leftIsPossible || upIsPossible;
                    if (isMovePossible) {
                        // printf("%d %d %d %d\n", rightIsPossible, downIsPossible, leftIsPossible, upIsPossible);
                    }
                }
                // rightIsPossible = borderLinks.x >= 1;
                // downIsPossible = borderLinks.y >= 1;
                // leftIsPossible = borderLinks.z >= 1;
                // upIsPossible = borderLinks.w >= 1;
            }

            char isMovePossible = rightIsPossible || downIsPossible || leftIsPossible || upIsPossible;
            unsigned int nextBorderLinkIdxRight = borderLinkIdx + 1;
            unsigned int nextBorderLinkIdxDown = borderLinkIdx + width;
            unsigned int nextBorderLinkIdxLeft = borderLinkIdx - 1;
            unsigned int nextBorderLinkIdxUp = borderLinkIdx - width;

            if (rightIsPossible && (!hasVisited[nextBorderLinkIdxRight] || state == 1)) {
                lastDir = 0;
                borderLinkIdx = borderLinkIdx + 1;
                borderLinks = links[borderLinkIdx];
            } else if (downIsPossible && (!hasVisited[nextBorderLinkIdxDown] || state == 1)) {
                lastDir = 1;
                borderLinkIdx = borderLinkIdx + width;
                borderLinks = links[borderLinkIdx];
            } else if (leftIsPossible && (!hasVisited[nextBorderLinkIdxLeft] || state == 1)) {
                lastDir = 2;
                borderLinkIdx = borderLinkIdx - 1;
                borderLinks = links[borderLinkIdx];
            } else if ((upIsPossible && !hasVisited[nextBorderLinkIdxUp] || state == 1)) {
                lastDir = 3;
                borderLinkIdx = borderLinkIdx - width;
                borderLinks = links[borderLinkIdx];
            } else {
                // state = 1;
                // printf("jun");
                done = true;
            }
            if (borderLinkIdx >= firstBorderLinkIdx) {
                hasVisited[firstBorderLinkIdx] = 0;
                hasVisited[borderLinkIdx] = 0;
                break;
            }

            // printf("doin iter %d \n", iters);
            // labels[borderLinkIdx] = 
            atomicMax(&labels[borderLinkIdx], rootCandidateLabel);

            // if (iters >= 1000) {
            //     done = true;
            // }

            // unsigned int hasVisited = 0x40000000; 


            // if (rightIsPossible) {
            //     lastDir = 0;
            //     borderLinkIdx = borderLinkIdx + 1;
            //     borderLinks = links[borderLinkIdx];
            // } else {
            //     // junction, dead end, connection
            //     lastDir = 255;
            // }

            // if (!isMovePossible) {
            //     // junction, dead end
            //     lastDir = 255;
            // }

            // if (rightIsPossible) {
            //     lastDir = 0;
            //     borderLinkIdx = borderLinkIdx + 1;
            //     borderLinks = links[borderLinkIdx];
            // } else if (downIsPossible) {
            //     lastDir = 1;
            //     borderLinkIdx = borderLinkIdx + width;
            //     borderLinks = links[borderLinkIdx];
            // } else if (leftIsPossible) {
            //     lastDir = 2;
            //     borderLinkIdx = borderLinkIdx - 1;
            //     borderLinks = links[borderLinkIdx];
            // } else if (upIsPossible) {
            //     lastDir = 3;
            //     borderLinkIdx = borderLinkIdx - width;
            //     borderLinks = links[borderLinkIdx];
            // } else {
            //     printf("jun");
            //     done = true;
            //     // lastDir = 255;
            // }     

            // if (iters > 5000) {
            //     done = true;
            // }
            // labels[borderLinkIdx] = 233550;
            
            iters += 1;

            hasVisited[borderLinkIdx] = 1;
        }

        // labels[firstBorderLinkIdx] = 233550;
    }

    __global__ void fetchFromBorder(unsigned int* labels, ushort4* links, int width, int height) {
        unsigned int c = blockIdx.x * blockDim.x + threadIdx.x;
        unsigned int r = blockIdx.y * blockDim.y + threadIdx.y;

        if (c >= width || r >= height) {
            return;
        }

        int outIdx = r * width + c;
        unsigned int currentLabel = labels[outIdx];

        ushort4 currentLinks = links[outIdx];
        unsigned int mask = 0b01111111111111111111111111111111;
        
        unsigned int farDownLabel = labels[(r + currentLinks.y) * width + c] & mask;
        unsigned int farRightLabel = labels[(r) * width + c + currentLinks.x] & mask;
        unsigned int farLeftLabel = labels[(r) * width + c - currentLinks.z] & mask;
        unsigned int farUpLabel = labels[(r - currentLinks.w) * width + c] & mask;

        labels[outIdx] = max(max(max(farRightLabel, farUpLabel), farDownLabel), farLeftLabel);
        // labels[outIdx] = max(max(max(maybeRootViaDown, farUpLabel), farDownLabel), farLeftLabel);
    }
    
}