
#include <hip/hip_runtime.h>
extern "C" {
    __global__ void labelWithSharedLinks(unsigned int* labels, ushort4* links, unsigned char* R,unsigned char* G,unsigned char* B, int width, int height) {
        unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
        unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
        if (x >= width || y >= height) {
            return;
        } 
        int threshold = 20;
        
        unsigned int connections = 0;

        uchar4 currentPixel = make_uchar4(R[y * width + x], G[y * width + x], B[y * width + x], 255);

        
        __shared__ ushort4 sharedLinks[32][32];
        ushort4 currentLink = make_ushort4(0, 0, 0, 0);

        
        // right 
        if (x < width-1) {
            uchar4 rightPixel = make_uchar4(R[y * width + x +1], G[y * width + x +1], B[y * width + x+1], 255);
            // uchar4 rightPixel = img[y * width + x + 1];
             
            if (abs(rightPixel.x - currentPixel.x) < threshold && abs(rightPixel.y - currentPixel.y) < threshold && abs(rightPixel.z - currentPixel.z) < threshold) {    
                currentLink.x = 1;
                connections |= (1u << 31); 
            }
        }
        // left
        if (x > 0) {
            uchar4 leftPixel = make_uchar4(R[y * width + x -1], G[y * width + x -1], B[y * width + x-1], 255); 
            // uchar4 leftPixel = img[y * width + x - 1];

          if (abs(leftPixel.x - currentPixel.x) < threshold && abs(leftPixel.y - currentPixel.y) < threshold && abs(leftPixel.z - currentPixel.z) < threshold) {    
                currentLink.z = 1;
                connections |= (1u << 30); 
            }
        }

        if (y < height -1) { 
            // down 
            uchar4 downPixel = make_uchar4(R[(y+1) * width + x ], G[(y +1) * width + x], B[(y+ 1) * width + x], 255);
            // uchar4 downPixel = img[(y + 1) * width + x];

            if (abs(downPixel.x - currentPixel.x) < threshold && abs(downPixel.y - currentPixel.y) < threshold && abs(downPixel.z - currentPixel.z) < threshold) {    
                currentLink.y = 1;
                connections |= (1u << 29); 
            }
        }
        if (y > 0) { 
            // up
            uchar4 upPixel = make_uchar4(R[(y-1) * width + x ], G[(y -1) * width + x], B[(y- 1) * width + x], 255);
            // uchar4 upPixel = img[(y - 1) * width + x];

            if (abs(upPixel.x - currentPixel.x) < threshold && abs(upPixel.y - currentPixel.y) < threshold && abs(upPixel.z - currentPixel.z) < threshold) {    
                currentLink.w = 1;
                connections |= (1u << 28);
            }
        }

        unsigned int labelIdx = y * width + x;
        unsigned int label = labelIdx + 1;

        sharedLinks[threadIdx.y][threadIdx.x] = currentLink;
        __syncthreads();

        for (int i=0; i<6; i++) {
            if (threadIdx.x + currentLink.x < 32) {
                // right
                currentLink.x += sharedLinks[threadIdx.y][threadIdx.x + currentLink.x].x;
            }

            if (threadIdx.y + currentLink.y < 32) {
                // down
                currentLink.y += sharedLinks[threadIdx.y + currentLink.y][threadIdx.x].y;
            }

            if ((int)threadIdx.x - (int)currentLink.z >= 0) {
                // left
                currentLink.z += sharedLinks[threadIdx.y][threadIdx.x - currentLink.z].z;
            }

            if ((int)threadIdx.y - (int)currentLink.w >= 0) {
                // up
                currentLink.w += sharedLinks[threadIdx.y - currentLink.w][threadIdx.x].w;
            }

            sharedLinks[threadIdx.y][threadIdx.x] = currentLink;
            __syncthreads();
        }

        links[labelIdx] = sharedLinks[threadIdx.y][threadIdx.x];

        
        // links[labelIdx] = currentLink;

        labels[labelIdx] = label;
    }

    __global__ void globalizeLinks(ushort4* links, int active_xr, int active_xl, int width, int height) {
        unsigned int xr = active_xr * blockDim.x + threadIdx.x;
        unsigned int xl = active_xl * blockDim.x + threadIdx.x;
        unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
        if (y >= height) {
            return;
        } 

        // if (xl < width) {
        //     unsigned short acc_link_z = links[y * width + xl].z;
        //     unsigned short leftMove = acc_link_z;

        //     while (leftMove != 0) {
        //         leftMove = links[y * width + xl - acc_link_z].z;
        //         acc_link_z += leftMove;
        //     }
        //     links[y * width + xl].z = acc_link_z;
        // }

        if (xr < width) {
            unsigned short acc_link_x = links[y * width + xr].x;
            unsigned short rightMove = acc_link_x;

            while (rightMove != 0) {
                rightMove = links[y * width + xr + acc_link_x].x;
                acc_link_x += rightMove;
            }
            links[y * width + xr].x = acc_link_x;
        }
    }

    __global__ void labelWithConnectionInfoMore32(unsigned int* labels, ushort4* links, unsigned char* R,unsigned char* G,unsigned char* B, int cycles, int width, int height) {
        unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
        unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
        if (x >= width || y >= height) {
            return;
        } 
        int threshold = 20;
        
        unsigned int connections = 0;

        uchar4 currentPixel = make_uchar4(R[y * width + x], G[y * width + x], B[y * width + x], 255);

        
        ushort4 currentLink = make_ushort4(0, 0, 0, 0);

        
        // right 
        if (x < width-1) {
            uchar4 rightPixel = make_uchar4(R[y * width + x +1], G[y * width + x +1], B[y * width + x+1], 255);
            // uchar4 rightPixel = img[y * width + x + 1];
             
            if (abs(rightPixel.x - currentPixel.x) < threshold && abs(rightPixel.y - currentPixel.y) < threshold && abs(rightPixel.z - currentPixel.z) < threshold) {    
                currentLink.x = 1;
                connections |= (1u << 31); 
            }
        }
        // left
        if (x > 0) {
            uchar4 leftPixel = make_uchar4(R[y * width + x -1], G[y * width + x -1], B[y * width + x-1], 255); 
            // uchar4 leftPixel = img[y * width + x - 1];

          if (abs(leftPixel.x - currentPixel.x) < threshold && abs(leftPixel.y - currentPixel.y) < threshold && abs(leftPixel.z - currentPixel.z) < threshold) {    
                currentLink.z = 1;
                connections |= (1u << 30); 
            }
        }

        if (y < height -1) { 
            // down 
            uchar4 downPixel = make_uchar4(R[(y+1) * width + x ], G[(y +1) * width + x], B[(y+ 1) * width + x], 255);
            // uchar4 downPixel = img[(y + 1) * width + x];

            if (abs(downPixel.x - currentPixel.x) < threshold && abs(downPixel.y - currentPixel.y) < threshold && abs(downPixel.z - currentPixel.z) < threshold) {    
                currentLink.y = 1;
                connections |= (1u << 29); 
            }
        }
        if (y > 0) { 
            // up
            uchar4 upPixel = make_uchar4(R[(y-1) * width + x ], G[(y -1) * width + x], B[(y- 1) * width + x], 255);
            // uchar4 upPixel = img[(y - 1) * width + x];

            if (abs(upPixel.x - currentPixel.x) < threshold && abs(upPixel.y - currentPixel.y) < threshold && abs(upPixel.z - currentPixel.z) < threshold) {    
                currentLink.w = 1;
                connections |= (1u << 28);
            }
        }

        unsigned int labelIdx = y * width + x;
        unsigned int label = labelIdx + 1;
        // label |= connections;

        // right
        for (int i = x + 1; i < width; i++) {
            // break; 
            int4 rightPixel = make_int4(R[y * width + i], G[y * width + i], B[y * width + i], 255);
            if (!(abs(rightPixel.x - currentPixel.x) < threshold && abs(rightPixel.y - currentPixel.y) < threshold && abs(rightPixel.z - currentPixel.z) < threshold)) {
                break;
            }
            unsigned short farRightLink = (unsigned short) (i - x);
            // printf("right link: %d \n", farRightLink);
            currentLink.x = farRightLink;
        }

        // down 
        for (int i = y + 1; i < height; i++) { 
            // break;
            int4 rightPixel = make_int4(R[i * width + x], G[i * width + x], B[i * width + x], 255);
            // printf("%d \n", rightPixel.x);
            if (!(abs(rightPixel.x - currentPixel.x) < threshold && abs(rightPixel.y - currentPixel.y) < threshold && abs(rightPixel.z - currentPixel.z) < threshold)) {
                break;
            }
            unsigned short farDownLabel = (unsigned short) (i - y);
            currentLink.y = farDownLabel;
        }
        // left
        for (int i = x - 1; i >= 0; i--) {
            // break; 
            int4 leftPixel = make_int4(R[y * width + i], G[y * width + i], B[y * width + i], 255);
            if (!(abs(leftPixel.x - currentPixel.x) < threshold && abs(leftPixel.y - currentPixel.y) < threshold && abs(leftPixel.z - currentPixel.z) < threshold)) {
                break;
            }
            unsigned short farLeftLabel = (unsigned short) (x - i);
            // printf("right link: %d \n", farLeftLabel);
            currentLink.z = farLeftLabel;
        }
        // up 
        for (int i = y - 1; i >= 0; i--) { 
            // break;
            int4 rightPixel = make_int4(R[i * width + x], G[i * width + x], B[i * width + x], 255);
            // printf("%d \n", rightPixel.x);
            if (!(abs(rightPixel.x - currentPixel.x) < threshold && abs(rightPixel.y - currentPixel.y) < threshold && abs(rightPixel.z - currentPixel.z) < threshold)) {
                break;
            }
            unsigned short farDownLabel = (unsigned short) (y - i);
            currentLink.w = farDownLabel;
        }

        links[labelIdx] = currentLink;

        // printf("label: %u \n", label);
        labels[labelIdx] = label;

    }

    __global__ void labelComponentsFar(unsigned int* input, unsigned int* out, ushort4* links, int width, int height, int* hasUpdated) {
        // return;
        unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
        unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x >= width || y >= height) {
            return;
        }

        int outIdx = y * width + x;
        
        unsigned int currentLabel = input[outIdx];

        ushort4 currentLink = links[outIdx];
        unsigned int farRightLabel = input[outIdx + (int) currentLink.x];

        if (farRightLabel > currentLabel) {
            // out[outIdx] = farRightLabel;
            currentLabel = farRightLabel;
            *hasUpdated = 1;
            // atomicOr(hasUpdated, 1);
            // return;
        }
    
        unsigned int farDownLabel = input[(y + currentLink.y) * width + x];

        if (farDownLabel > currentLabel) {
            // out[outIdx] = farDownLabel;
            currentLabel = farDownLabel;
            *hasUpdated = 1;
            // atomicOr(hasUpdated, 1);
            // return;
        }    

        unsigned int farLeftLabel = input[outIdx - currentLink.z];

        if (farLeftLabel > currentLabel) {
            // out[outIdx] = farLeftLabel;
            currentLabel = farLeftLabel;
            *hasUpdated = 1;
            // atomicOr(hasUpdated, 1);
            // return;
        }
     
        
        unsigned int farUpLabel = input[(y - currentLink.w) * width + x];

        if (farUpLabel > currentLabel) {
            // out[outIdx] = farUpLabel;
            currentLabel = farUpLabel;
            *hasUpdated = 1;
            // atomicOr(hasUpdated, 1);
            // return;
        }
        // if (outIdx == 0) {
        //     printf("%d \n", (currentLabel >> 30) & 1);
        // }
        
        int leftLabel = input[outIdx - min(1, currentLink.z)];

        if (leftLabel > currentLabel) {
            // out[outIdx] = farLeftLabel;
            currentLabel = leftLabel;
            *hasUpdated = 1;
        }
  
        int upLabel = input[(y - min(1, currentLink.w)) * width + x];

        if (upLabel > currentLabel) {
            // out[outIdx] = farLeftLabel;
            currentLabel = upLabel;
            *hasUpdated = 1;
        }
 

        // int farUpLabel = input[(y - currentLink.w) * width + x];

        // if (farUpLabel > currentLabel) {
        //     // out[outIdx] = farUpLabel;
        //     currentLabel = farUpLabel;
        //     *hasUpdated = 1;
        //     // atomicOr(hasUpdated, 1);
        //     // return;
        // }
        out[outIdx] = currentLabel;
    }
 

}