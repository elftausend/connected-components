
#include <hip/hip_runtime.h>
extern "C" {
    

    __global__ void setRootLabelIter(ushort4* links, unsigned int* labels, unsigned char* rootCandidates, int width, int height) {
        unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
        unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
        if (x >= width || y >= height) {
            return;
        } 
        int outIdx = y * width + x;
        
        unsigned int currentLabel = labels[outIdx];

        ushort4 currentLink = links[outIdx];


        unsigned int farRightIdx = outIdx + (int) currentLink.x;
        unsigned int farDownIdx = (y + currentLink.y) * width + x;
        // unsigned int farLeftIdx = outIdx - currentLink.z;
        // unsigned int farUpIdx = (y - currentLink.w) * width + x;
        
        if (rootCandidates[farRightIdx]) {
            labels[outIdx] = labels[farRightIdx];
        }

        if (rootCandidates[farDownIdx]) {
            labels[outIdx] = labels[farDownIdx];
        }
    }

    __global__ void labelWithConnectionInfoMore32(unsigned int* labels, ushort4* links, unsigned char* R,unsigned char* G,unsigned char* B, int cycles, int width, int height) {
        unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
        unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
        if (x >= width || y >= height) {
            return;
        } 
        int threshold = 20;
        
        unsigned int connections = 0;

        uchar4 currentPixel = make_uchar4(R[y * width + x], G[y * width + x], B[y * width + x], 255);

        
        ushort4 currentLink = make_ushort4(0, 0, 0, 0);

        
        // right 
        if (x < width-1) {
            uchar4 rightPixel = make_uchar4(R[y * width + x +1], G[y * width + x +1], B[y * width + x+1], 255);
            // uchar4 rightPixel = img[y * width + x + 1];
             
            if (abs(rightPixel.x - currentPixel.x) < threshold && abs(rightPixel.y - currentPixel.y) < threshold && abs(rightPixel.z - currentPixel.z) < threshold) {    
                currentLink.x = 1;
                connections |= (1u << 31); 
            }
        }
        // left
        if (x > 0) {
            uchar4 leftPixel = make_uchar4(R[y * width + x -1], G[y * width + x -1], B[y * width + x-1], 255); 
            // uchar4 leftPixel = img[y * width + x - 1];

          if (abs(leftPixel.x - currentPixel.x) < threshold && abs(leftPixel.y - currentPixel.y) < threshold && abs(leftPixel.z - currentPixel.z) < threshold) {    
                currentLink.z = 1;
                connections |= (1u << 30); 
            }
        }

        if (y < height -1) { 
            // down 
            uchar4 downPixel = make_uchar4(R[(y+1) * width + x ], G[(y +1) * width + x], B[(y+ 1) * width + x], 255);
            // uchar4 downPixel = img[(y + 1) * width + x];

            if (abs(downPixel.x - currentPixel.x) < threshold && abs(downPixel.y - currentPixel.y) < threshold && abs(downPixel.z - currentPixel.z) < threshold) {    
                currentLink.y = 1;
                connections |= (1u << 29); 
            }
        }
        if (y > 0) { 
            // up
            uchar4 upPixel = make_uchar4(R[(y-1) * width + x ], G[(y -1) * width + x], B[(y- 1) * width + x], 255);
            // uchar4 upPixel = img[(y - 1) * width + x];

            if (abs(upPixel.x - currentPixel.x) < threshold && abs(upPixel.y - currentPixel.y) < threshold && abs(upPixel.z - currentPixel.z) < threshold) {    
                currentLink.w = 1;
                connections |= (1u << 28);
            }
        }

        unsigned int labelIdx = y * width + x;
        unsigned int label = labelIdx + 1;
        // label |= connections;

        // right
        for (int i = x + 1; i < width; i++) {
            // break; 
            int4 rightPixel = make_int4(R[y * width + i], G[y * width + i], B[y * width + i], 255);
            if (!(abs(rightPixel.x - currentPixel.x) < threshold && abs(rightPixel.y - currentPixel.y) < threshold && abs(rightPixel.z - currentPixel.z) < threshold)) {
                break;
            }
            unsigned short farRightLink = (unsigned short) (i - x);
            // printf("right link: %d \n", farRightLink);
            currentLink.x = farRightLink;
        }

        // down 
        for (int i = y + 1; i < height; i++) { 
            // break;
            int4 rightPixel = make_int4(R[i * width + x], G[i * width + x], B[i * width + x], 255);
            // printf("%d \n", rightPixel.x);
            if (!(abs(rightPixel.x - currentPixel.x) < threshold && abs(rightPixel.y - currentPixel.y) < threshold && abs(rightPixel.z - currentPixel.z) < threshold)) {
                break;
            }
            unsigned short farDownLabel = (unsigned short) (i - y);
            currentLink.y = farDownLabel;
        }
        // left
        for (int i = x - 1; i >= 0; i--) {
            // break; 
            int4 leftPixel = make_int4(R[y * width + i], G[y * width + i], B[y * width + i], 255);
            if (!(abs(leftPixel.x - currentPixel.x) < threshold && abs(leftPixel.y - currentPixel.y) < threshold && abs(leftPixel.z - currentPixel.z) < threshold)) {
                break;
            }
            unsigned short farLeftLabel = (unsigned short) (x - i);
            // printf("right link: %d \n", farLeftLabel);
            currentLink.z = farLeftLabel;
        }
        // up 
        for (int i = y - 1; i >= 0; i--) { 
            // break;
            int4 rightPixel = make_int4(R[i * width + x], G[i * width + x], B[i * width + x], 255);
            // printf("%d \n", rightPixel.x);
            if (!(abs(rightPixel.x - currentPixel.x) < threshold && abs(rightPixel.y - currentPixel.y) < threshold && abs(rightPixel.z - currentPixel.z) < threshold)) {
                break;
            }
            unsigned short farDownLabel = (unsigned short) (y - i);
            currentLink.w = farDownLabel;
        }

        links[labelIdx] = currentLink;

        // printf("label: %u \n", label);
        labels[labelIdx] = label;

    }
    
    
    // could use bit shifting => store root bit in label
    __global__ void classifyRootCandidates(unsigned int* input, ushort4* links, unsigned char* rootCandidates, int width, int height) {
        unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
        unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x >= width || y >= height) {
            return;
        }

        int outIdx = y * width + x;
        
        unsigned int currentLabel = input[outIdx];
        ushort4 currentLink = links[outIdx];

        // if (currentLink.x == 0 && currentLink.y == 0) {
        //     rootCandidates[outIdx] = 1;
        // }
        unsigned int farRightLabel = input[outIdx + (int) currentLink.x];
        unsigned int farDownLabel = input[(y + currentLink.y) * width + x];

        if (farRightLabel > currentLabel || farDownLabel > currentLabel) {
            rootCandidates[outIdx] = 0;
            return;
        }
        // could use bit shifting => store root bit in label
        rootCandidates[outIdx] = 1;
    }

    __device__ void setRootLinkIfCandidate(unsigned int maybe_root_link_idx, unsigned int currentIdx, unsigned int* rootLinks, unsigned char* rootCandidates) {
        if (rootCandidates[maybe_root_link_idx]) {
            rootLinks[currentIdx] = maybe_root_link_idx;
        }
    }

    
    __global__ void labelComponentsFar(unsigned int* input, unsigned int* out, ushort4* links, int width, int height, int* hasUpdated) {
        // return;
        unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
        unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x >= width || y >= height) {
            return;
        }

        int outIdx = y * width + x;
        
        unsigned int currentLabel = input[outIdx];

        ushort4 currentLink = links[outIdx];
        unsigned int farRightLabel = input[outIdx + (int) currentLink.x];

        if (farRightLabel > currentLabel) {
            // out[outIdx] = farRightLabel;
            currentLabel = farRightLabel;
            *hasUpdated = 1;
            // atomicOr(hasUpdated, 1);
            // return;
        }
    
        unsigned int farDownLabel = input[(y + currentLink.y) * width + x];

        if (farDownLabel > currentLabel) {
            // out[outIdx] = farDownLabel;
            currentLabel = farDownLabel;
            *hasUpdated = 1;
            // atomicOr(hasUpdated, 1);
            // return;
        }    

        unsigned int farLeftLabel = input[outIdx - currentLink.z];

        if (farLeftLabel > currentLabel) {
            // out[outIdx] = farLeftLabel;
            currentLabel = farLeftLabel;
            *hasUpdated = 1;
            // atomicOr(hasUpdated, 1);
            // return;
        }
     
        
        unsigned int farUpLabel = input[(y - currentLink.w) * width + x];

        if (farUpLabel > currentLabel) {
            // out[outIdx] = farUpLabel;
            currentLabel = farUpLabel;
            *hasUpdated = 1;
            // atomicOr(hasUpdated, 1);
            // return;
        }
        // if (outIdx == 0) {
        //     printf("%d \n", (currentLabel >> 30) & 1);
        // }
        
        int leftLabel = input[outIdx - min(1, currentLink.z)];

        if (leftLabel > currentLabel) {
            // out[outIdx] = farLeftLabel;
            currentLabel = leftLabel;
            *hasUpdated = 1;
        }
  
        int upLabel = input[(y - min(1, currentLink.w)) * width + x];

        if (upLabel > currentLabel) {
            // out[outIdx] = farLeftLabel;
            currentLabel = upLabel;
            *hasUpdated = 1;
        }
 

        // int farUpLabel = input[(y - currentLink.w) * width + x];

        // if (farUpLabel > currentLabel) {
        //     // out[outIdx] = farUpLabel;
        //     currentLabel = farUpLabel;
        //     *hasUpdated = 1;
        //     // atomicOr(hasUpdated, 1);
        //     // return;
        // }
        out[outIdx] = currentLabel;
    }
    
    __global__ void initRootLinks(unsigned int* rootLinks, int width, int height) {
        unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
        unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x >= width || y >= height) {
            return;
        }

        int outIdx = y * width + x;
        rootLinks[outIdx] = outIdx;

    }

}