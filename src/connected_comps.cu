
#include <hip/hip_runtime.h>
extern "C"{
    
    __global__ void writeToSurface(hipSurfaceObject_t target, int width, int height, char r, char g, char b) {
        unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
        unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    
        if (x < width && y < height) {
            uchar4 data = make_uchar4(r, g, b, 0xff);
            surf2Dwrite(data, target, x * sizeof(uchar4), y);
        }
    }
    
    __global__ void interleaveRGB(hipSurfaceObject_t target, int width, int height,
            unsigned char *R, unsigned char *G, unsigned char *B )
    {
        unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
        unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

        if(x < width && y < height) {       
            unsigned char valR = R[y * width + x]; 
            unsigned char valG = G[y * width + x]; 
            unsigned char valB = B[y * width + x]; 
            uchar4 data = make_uchar4(valR, valG, valB, 0xff);
            surf2Dwrite(data, target, x * sizeof(uchar4), height -1- y);
        }
    }

    __global__ void startlabelComponents(hipSurfaceObject_t target, int width, int height) {
        unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
        unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    
        if (!(x < width && y < height)) {
            return;
        }

        float normX = (float) x / (float) width;
        int colorX = (unsigned char) (normX * 255.0f);

        float normY = (float) y / (float) height;
        int colorY = (unsigned char) (normY * 255.0f);

        uchar4 color = make_uchar4(colorX, colorY, 0, 255);

        surf2Dwrite(color, target, x * sizeof(uchar4), height -1 - y);
    }

    __global__ void computeLabels(hipTextureObject_t input, hipSurfaceObject_t out, int width, int height) {
        unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
        unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    
        if (!(x < width && y < height)) {
            return;
        }

        int threshold = 10;

//        surf2Dwrite(color, target, x * sizeof(uchar4), height -1 - y);
    }
}