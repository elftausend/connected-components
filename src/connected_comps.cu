
#include <hip/hip_runtime.h>
extern "C"{
    
    __global__ void writeToSurface(hipSurfaceObject_t target, int width, int height, char r, char g, char b) {
        unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
        unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    
        if (x < width && y < height) {
            uchar4 data = make_uchar4(r, g, b, 0xff);
            surf2Dwrite(data, target, x * sizeof(uchar4), y);
        }
    }
    
    __global__ void interleaveRGB(hipSurfaceObject_t target, int width, int height,
            unsigned char *R, unsigned char *G, unsigned char *B )
    {
        unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
        unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

        if(x < width && y < height) {       
            unsigned char valR = R[y * width + x]; 
            unsigned char valG = G[y * width + x]; 
            unsigned char valB = B[y * width + x]; 
            uchar4 data = make_uchar4(valR, valG, valB, 0xff);
            surf2Dwrite(data, target, x * sizeof(uchar4), height -1- y);
        }
    }

    __global__ void labelComponents(uchar4* target, int width, int height) {
        unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
        unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    
        if (!(x < width && y < height)) {
            return;
        }

        float normX = (float) x / (float) width;
        int colorX = (unsigned char) (normX * 255.0f);

        float normY = (float) y / (float) height;
        int colorY = (unsigned char) (normY * 255.0f);

        uchar4 color = make_uchar4(colorX, colorY, 0, 255);

        target[y * width + x] = color;

        //surf2Dwrite(color, target, x * sizeof(uchar4), height -1 - y);
    }

    __global__ void computeLabels(uchar4* input, hipSurfaceObject_t out, int width, int height, unsigned char *R, unsigned char *G, unsigned char *B) {
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x >= width) {
            return;
        }
        
        if (y >= height) {
            return;
        }
        

        // shared?
        unsigned char valR = R[y * width + x]; 
        unsigned char valG = G[y * width + x]; 
        unsigned char valB = B[y * width + x]; 

        int threshold = 4;
        uchar4 currentLabel = input[y * width + x];

        if (x+1 < width) {
            unsigned char valRedRight = R[y * width + x + 1];
            unsigned char valGreenRight = G[y * width + x + 1];
            unsigned char valBlueRight = B[y * width + x + 1];
            uchar4 labelRight = input[y * width + x + 1];
            if (abs(valRedRight - valR) < threshold && abs(valGreenRight - valG) < threshold && abs(valBlueRight - valB) < threshold) {
                
                if ((int) currentLabel.x + (int) currentLabel.y < (int) labelRight.x + (int) labelRight.y) {
                    if (blockIdx.y == 0) {
                        printf("label right: %d %d %d\n", labelRight.x, labelRight.y, labelRight.z);
                    }
                    surf2Dwrite(labelRight, out, x * sizeof(char4), height -1 - y);
                    return;
                }
            }   
            surf2Dwrite(currentLabel, out, x * sizeof(char4), height -1 - y);
        }
    }
}