
#include <hip/hip_runtime.h>
extern "C"{
    
    __global__ void writeToSurface(hipSurfaceObject_t target, int width, int height, char r, char g, char b) {
        unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
        unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    
        if (x < width && y < height) {
            uchar4 data = make_uchar4(r, g, b, 0xff);
            surf2Dwrite(data, target, x * sizeof(uchar4), y);
        }
    }
    
    __global__ void interleaveRGB(hipSurfaceObject_t target, int width, int height,
            unsigned char *R, unsigned char *G, unsigned char *B )
    {
        unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
        unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

        if(x < width && y < height) {       
            unsigned char valR = R[y * width + x]; 
            unsigned char valG = G[y * width + x]; 
            unsigned char valB = B[y * width + x]; 
            uchar4 data = make_uchar4(valR, valG, valB, 0xff);
            surf2Dwrite(data, target, x * sizeof(uchar4), height -1- y);
        }
    }

    __global__ void labelPixels(uchar4* target, int width, int height) {
        unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
        unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    
        if (!(x < width && y < height)) {
            return;
        }

        float normX = (float) x / (float) width;
        unsigned char colorX = (unsigned char) (normX * 255.0f);

        float normY = (float) y / (float) height;
        unsigned char colorY = (unsigned char) (normY * 255.0f);

        // float normZ = (float) (x + y) / (float) (width + height);
        // int colorZ = (unsigned char) (normZ * 255.0f);
        unsigned char colorZ = 0;

        uchar4 color = make_uchar4(colorX, colorY, colorZ, 255);

        target[y * width + x] = color;

        //surf2Dwrite(color, target, x * sizeof(uchar4), height -1 - y);
    }

    __global__ void labelPixelsCombinations(uchar4* target, int width, int height) {
        unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
        unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    
        if (!(x < width && y < height)) {
            return;
        }

        // unsigned char colorZ = (unsigned char) (x + y) / (255*255);

        unsigned char colorZ = x / 255 + (y / 255) * ((width) / 255) + (y / 255);

        uchar4 color = make_uchar4(x % 255, y % 255, colorZ, 255);
        // printf("color: %d %d %d\n", color.x, color.y, color.z);

        target[y * width + x] = color;

        //surf2Dwrite(color, target, x * sizeof(uchar4), height -1 - y);
    }

    __global__ void labelPixelsRowed(uchar4* target, int width, int height) {
        unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
        unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    
        if (!(x < width && y < height)) {
            return;
        }

        float normX = (float) x / (float) width;
        int colorX = (unsigned char) (normX * 255.0f);

        uchar4 color = make_uchar4(colorX, 0, 0, 255);

        target[y * width + x] = color;
    }

    __global__ void labelComponents(uchar4* input, uchar4* out, int width, int height, unsigned char *R, unsigned char *G, unsigned char *B) {
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x >= width) {
            return;
        }
        
        if (y >= height) {
            return;
        }
        

        // shared?
        unsigned char valR = R[y * width + x]; 
        unsigned char valG = G[y * width + x]; 
        unsigned char valB = B[y * width + x]; 

        int threshold = 20;
        uchar4 currentLabel = input[y * width + x];

        if (x+1 < width) {
            unsigned char valRedRight = R[y * width + x + 1];
            unsigned char valGreenRight = G[y * width + x + 1];
            unsigned char valBlueRight = B[y * width + x + 1];
            uchar4 labelRight = input[y * width + x + 1];
            
            if (abs(valRedRight - valR) < threshold && abs(valGreenRight - valG) < threshold && abs(valBlueRight - valB) < threshold) {
                
                if ((int) currentLabel.x + (int) currentLabel.y + (int) currentLabel.z < (int) labelRight.x + (int) labelRight.y + (int) labelRight.z) {
                    if (blockIdx.y == 0) {
                        // printf("label right: %d %d %d\n", labelRight.x, labelRight.y, labelRight.z);
                    }
                    // currentLabel = labelRight;
                    out[y * width + x] = labelRight;
                    //surf2Dwrite(labelRight, out, x * sizeof(char4), height -1 - y);
                    return;
                }
            }   
            //surf2Dwrite(currentLabel, out, x * sizeof(char4), height -1 - y);
        }

        if (x-1 < width) {
            unsigned char valRedRight = R[y * width + x - 1];
            unsigned char valGreenRight = G[y * width + x - 1];
            unsigned char valBlueRight = B[y * width + x - 1];
            uchar4 labelRight = input[y * width + x - 1];
            if (abs(valRedRight - valR) < threshold && abs(valGreenRight - valG) < threshold && abs(valBlueRight - valB) < threshold) {
                
                if ((int) currentLabel.x + (int) currentLabel.y + (int) currentLabel.z < (int) labelRight.x + (int) labelRight.y + (int) labelRight.z) {
                    // currentLabel = labelRight;
                    out[y * width + x] = labelRight;
                    return;
                }
            }   
        }

        if (y+1 < height) {
            unsigned char valRedBottom = R[(y+1) * width + x];
            unsigned char valGreenBottom = G[(y+1) * width + x];
            unsigned char valBlueBottom = B[(y+1) * width + x];
            uchar4 labelBottom = input[(y+1) * width + x];
            if (abs(valRedBottom - valR) < threshold && abs(valGreenBottom - valG) < threshold && abs(valBlueBottom - valB) < threshold) {
                if ((int) currentLabel.x + (int) currentLabel.y + (int) currentLabel.z < (int) labelBottom.x + (int) labelBottom.y + (int) labelBottom.z) {
                    // currentLabel = labelBottom;
                    out[y * width + x] = labelBottom;
                    return;
                }
            }
        }
        if (y-1 < height) {
            unsigned char valRedBottom = R[(y-1) * width + x];
            unsigned char valGreenBottom = G[(y-1) * width + x];
            unsigned char valBlueBottom = B[(y-1) * width + x];
            uchar4 labelBottom = input[(y-1) * width + x];
            if (abs(valRedBottom - valR) < threshold && abs(valGreenBottom - valG) < threshold && abs(valBlueBottom - valB) < threshold) {
                if ((int) currentLabel.x + (int) currentLabel.y + (int) currentLabel.z < (int) labelBottom.x + (int) labelBottom.y + (int) labelBottom.z) {
                    // currentLabel = labelBottom;
                    out[y * width + x] = labelBottom;
                    return;
                }
            }
        }

        out[y * width + x] = currentLabel;
    }

    __global__ void labelComponentsRowed(uchar4* input, uchar4* out, int width, int height, unsigned char *R, unsigned char *G, unsigned char *B) {
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x >= width) {
            return;
        }
        
        if (y >= height) {
            return;
        }

        // shared?
        unsigned char valR = R[y * width + x]; 
        unsigned char valG = G[y * width + x]; 
        unsigned char valB = B[y * width + x]; 

        int threshold = 30;
        uchar4 currentLabel = input[y * width + x];

        if (x+1 < width) {
            unsigned char valRedRight = R[y * width + x + 1];
            unsigned char valGreenRight = G[y * width + x + 1];
            unsigned char valBlueRight = B[y * width + x + 1];
            uchar4 labelRight = input[y * width + x + 1];
            
            if (abs(valRedRight - valR) < threshold && abs(valGreenRight - valG) < threshold && abs(valBlueRight - valB) < threshold) {
                if ((int) currentLabel.x + (int) currentLabel.y + (int) currentLabel.z < (int) labelRight.x + (int) labelRight.y + (int) labelRight.z) {
                    out[y * width + x] = labelRight;
                    return;
                }
            }   
        }

        if (y+1 < height) {
            unsigned char valRedBottom = R[(y+1) * width + x];
            unsigned char valGreenBottom = G[(y+1) * width + x];
            unsigned char valBlueBottom = B[(y+1) * width + x];
            uchar4 labelBottom = input[(y+1) * width + x];
            if (abs(valRedBottom - valR) < threshold && abs(valGreenBottom - valG) < threshold && abs(valBlueBottom - valB) < threshold) {
                if ((int) currentLabel.x + (int) currentLabel.y + (int) currentLabel.z < (int) labelBottom.x + (int) labelBottom.y + (int) labelBottom.z) {
                    out[y * width + x] = labelBottom;
                    return;
                }
            }
        }

        out[y * width + x] = currentLabel;
    }

    __global__ void copyToSurface(uchar4* input, hipSurfaceObject_t target, int width, int height) {
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x >= width) {
            return;
        }
        
        if (y >= height) {
            return;
        }
        
        uchar4 color = input[y * width + x];
        surf2Dwrite(color, target, x * sizeof(uchar4), height -1 - y);
    }

    __global__ void colorComponentAtPixel(hipTextureObject_t texture, hipSurfaceObject_t surface, int posX, int posY, int width, int height) {
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;

        __shared__ float4 toSearchLabel;
        if (threadIdx.x == 0) {
            toSearchLabel = tex2D<float4>(texture, posX, height -1 - posY);

        }
        __syncthreads();

        if (x >= width) {
            return;
        }
        
        if (y >= height) {
            return;
        }
        
        // uchar4 label = surf2Dread<uchar4>(surface, x * sizeof(uchar4), height -1 - y);
        float4 label = tex2D<float4>(texture, x, height -1 - y);

        float threshold = 0.019; // 0.019

        // mind that z is not used
        if (abs(toSearchLabel.x - label.x) < threshold && abs(toSearchLabel.y - label.y) < threshold && abs(toSearchLabel.z - label.z) < threshold) {
            uchar4 color = make_uchar4(0, 0, 255, 255);
            surf2Dwrite(color, surface, x * sizeof(uchar4), height -1 - y);
        }
        __syncthreads();

    }

    __global__ void colorComponentAtPixelExact(hipTextureObject_t texture, hipSurfaceObject_t surface, int posX, int posY, int width, int height, unsigned char R, unsigned char G, unsigned char B) {
        if (R == 0 && G == 0 && B == 255) {
            return;
        }
        
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x >= width) {
            return;
        }
        
        if (y >= height) {
            return;
        }

        uchar4 toSearchLabel = make_uchar4(R, G, B, 255);
        /*__shared__ uchar4 toSearchLabel;
        if (threadIdx.x == 0) {
            toSearchLabel = surf2Dread<uchar4>(surface, posX * sizeof(uchar4), height -1 - posY);
            // toSearchLabel = tex2D<uchar4>(texture, posX, posY);

        }
        __syncthreads();*/


        // uchar4 label = tex2D<uchar4>(texture, x, y);
        uchar4 label = surf2Dread<uchar4>(surface, x * sizeof(uchar4), height -1 - y);


        if (toSearchLabel.x == label.x && toSearchLabel.y == label.y && toSearchLabel.z == label.z) {
            uchar4 color = make_uchar4(0, 0, 255, 255);
            surf2Dwrite(color, surface, x * sizeof(uchar4), height -1 - y);
        }

    }

    __global__ void readPixelValue(uchar4* labels, int posX, int posY, unsigned char* R, unsigned char* G, unsigned char* B, int width, int height) {

        if (posX >= width) {
            return;
        }

        if (posY >= height) {
            return;
        }

        // uchar4 pixel = tex2D<uchar4>(texture, posX, posY);
        // uchar4 pixel = surf2Dread<uchar4>(surface, posX * sizeof(uchar4), height -1 -posY);
        uchar4 pixel = labels[posY * width + posX];
        R[0] = pixel.x;
        G[0] = pixel.y;
        B[0] = pixel.z;
    }
}