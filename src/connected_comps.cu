
#include <hip/hip_runtime.h>
extern "C"{
    
    __global__ void writeToSurface(hipSurfaceObject_t target, int width, int height, char r, char g, char b) {
        unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
        unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    
        if (x < width && y < height) {
            uchar4 data = make_uchar4(r, g, b, 0xff);
            surf2Dwrite(data, target, x * sizeof(uchar4), y);
        }
    }
    
    __global__ void interleaveRGB(hipSurfaceObject_t target, int width, int height,
            unsigned char *R, unsigned char *G, unsigned char *B )
    {
        unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
        unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

        if(x < width && y < height) {       
            unsigned char valR = R[y * width + x]; 
            unsigned char valG = G[y * width + x]; 
            unsigned char valB = B[y * width + x]; 
            uchar4 data = make_uchar4(valR, valG, valB, 0xff);
            surf2Dwrite(data, target, x * sizeof(uchar4), height -1- y);
        }
    }

    __global__ void labelPixels(uchar4* target, int width, int height) {
        unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
        unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    
        if (!(x < width && y < height)) {
            return;
        }

        float normX = (float) x / (float) width;
        unsigned char colorX = (unsigned char) (normX * 255.0f);

        float normY = (float) y / (float) height;
        unsigned char colorY = (unsigned char) (normY * 255.0f);

        // float normZ = (float) (x + y) / (float) (width + height);
        // int colorZ = (unsigned char) (normZ * 255.0f);
        unsigned char colorZ = 0;

        uchar4 color = make_uchar4(colorX, colorY, colorZ, 255);

        target[y * width + x] = color;

        //surf2Dwrite(color, target, x * sizeof(uchar4), height -1 - y);
    }

    __global__ void labelPixelsCombinations(uchar4* target, int width, int height) {
        unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
        unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    
        if (!(x < width && y < height)) {
            return;
        }

        // unsigned char colorZ = (unsigned char) (x + y) / (255*255);

        unsigned char colorZ = x / 255 + (y / 255) * ((width) / 255) + (y / 255);

        uchar4 color = make_uchar4(x % 255, y % 255, colorZ, 255);
        // printf("color: %d %d %d\n", color.x, color.y, color.z);

        target[y * width + x] = color;

        //surf2Dwrite(color, target, x * sizeof(uchar4), height -1 - y);
    }

    __global__ void labelPixelsRowed(uchar4* target, int width, int height) {
        unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
        unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    
        if (!(x < width && y < height)) {
            return;
        }

        float normX = (float) x / (float) width;
        int colorX = (unsigned char) (normX * 255.0f);

        uchar4 color = make_uchar4(colorX, 0, 0, 255);

        target[y * width + x] = color;
    }


    __global__ void labelComponentsShared(uchar4* input, uchar4* out, int width, int height, unsigned char* R, unsigned char* G, unsigned char* B, int threshold, unsigned char* hasUpdated, unsigned char offsetY, unsigned char offsetX) {
        int bloatedBlockIdxX = blockIdx.x * 2 + offsetX;
        int bloatedBlockIdxY = blockIdx.y * 2 + offsetY;

        int x = bloatedBlockIdxX * blockDim.x + threadIdx.x;
        int y = bloatedBlockIdxY * blockDim.y + threadIdx.y;

        if (x >= width) {
            return;
        }

        if (y >= height) {
            return;
        }

        /*unsigned char activeColor = blockIdx.x % 2 + blockIdx.y % 2 * 2;

        // printf("active color %d\n", activeColor);

        if (color != activeColor) {
            return;
        }*/

        __shared__ uchar4 pixels[34][34];
        __shared__ uchar4 labels[34][34];

        int newY = y;// - blockIdx.y;
        int newX = x;// - blockIdx.x;


        
        int outIdx = y * width + x;
        int pixelIdx = y * width + x;

        // int pixelIdx = y * width + x;
        // pixels[threadIdx.y][threadIdx.x] = make_uchar4(R[pixelIdx], G[pixelIdx], B[pixelIdx], 255);
        // labels[threadIdx.y][threadIdx.x] = input[(y-offsetY) * width + (x - offsetX)];
        // labels[threadIdx.y][threadIdx.x] = input[newY * width + newX];

        pixels[threadIdx.y+1][threadIdx.x+1] = make_uchar4(R[pixelIdx], G[pixelIdx], B[pixelIdx], 255);

        if (threadIdx.y == 0) {
            int upperOverlap = (bloatedBlockIdxY * blockDim.y -1);
            int upperOverlapIdx = upperOverlap * width + x;
            if (upperOverlap >= 0) {

                labels[0][threadIdx.x+1] = input[upperOverlapIdx];
                pixels[0][threadIdx.x+1] = make_uchar4(R[upperOverlapIdx], G[upperOverlapIdx], B[upperOverlapIdx], 255);
                if (threadIdx.x == 0) {
                    labels[0][0] = input[upperOverlapIdx - 1];
                    pixels[0][0] = make_uchar4(R[upperOverlapIdx - 1], G[upperOverlapIdx - 1], B[upperOverlapIdx - 1], 255);
                }
                if (threadIdx.x == 31) {
                    labels[0][33] = input[upperOverlapIdx + 1];
                    pixels[0][33] = make_uchar4(R[upperOverlapIdx + 1], G[upperOverlapIdx + 1], B[upperOverlapIdx + 1], 255);
                }
            }
        }

        if (threadIdx.y == 31) {
            int lowerOverlap =  (bloatedBlockIdxY * blockDim.y + 32);
            int lowerOverlapIdx = lowerOverlap * width + x;
            if (lowerOverlap < height) {                
                labels[33][threadIdx.x+1] = input[lowerOverlapIdx];
                pixels[33][threadIdx.x+1] = make_uchar4(R[lowerOverlapIdx], G[lowerOverlapIdx], B[lowerOverlapIdx], 255);
                if (threadIdx.x == 0) {
                    labels[33][0] = input[lowerOverlapIdx - 1];
                    pixels[33][0] = make_uchar4(R[lowerOverlapIdx - 1], G[lowerOverlapIdx - 1], B[lowerOverlapIdx - 1], 255);
                }
                if (threadIdx.x == 31) {
                    labels[33][33] = input[lowerOverlapIdx + 1];
                    pixels[33][33] = make_uchar4(R[lowerOverlapIdx + 1], G[lowerOverlapIdx + 1], B[lowerOverlapIdx + 1], 255);
                }
            }
        }

        if (threadIdx.x == 0) {
            int leftOverlap =  (bloatedBlockIdxX * blockDim.x -1);
            int leftOverlapIdx = y * width + leftOverlap;
            if (leftOverlap >= 0) {
                labels[threadIdx.y+1][0] = input[leftOverlapIdx];
                pixels[threadIdx.y+1][0] = make_uchar4(R[leftOverlapIdx], G[leftOverlapIdx], B[leftOverlapIdx], 255);
            }
        }

        if (threadIdx.x == 31) {
            int rightOverlap =  (bloatedBlockIdxX * blockDim.x + 32);
            int rightOverlapIdx = y * width + rightOverlap;
            if (rightOverlap < width) {
                labels[threadIdx.y+1][33] = input[rightOverlapIdx];
                pixels[threadIdx.y+1][33] = make_uchar4(R[rightOverlapIdx], G[rightOverlapIdx], B[rightOverlapIdx], 255);
            }
        }


        labels[threadIdx.y+1][threadIdx.x+1] = input[y * width + x];
        // labels[threadIdx.y - blockIdx.y][threadIdx.x - blockIdx.x] = input[y * width + x];        
        __syncthreads();

        uchar4 currentLabel = labels[threadIdx.y][threadIdx.x];
        uchar4 currentPixel = pixels[threadIdx.y][threadIdx.x];

        return;

        if (threadIdx.x+1 < 32) {
            uchar4 pixel = pixels[threadIdx.y][threadIdx.x + 1];
            uchar4 label = labels[threadIdx.y][threadIdx.x + 1];
            if (abs(pixel.x - currentPixel.x) < threshold && abs(pixel.y - currentPixel.y) < threshold && abs(pixel.z - currentPixel.z) < threshold) {    
                if ((int) currentLabel.x + (int) currentLabel.y + (int) currentLabel.z < (int) label.x + (int) label.y + (int) label.z) {
                    // labels[threadIdx.y][threadIdx.x] = label;
                    // __syncthreads();
                    hasUpdated[0] = 1; 
                    out[outIdx] = label;
                    return;
                }
            }
        }

        if (int (threadIdx.x)-1 > 0) {
            uchar4 pixel = pixels[threadIdx.y][threadIdx.x - 1];
            uchar4 label = labels[threadIdx.y][threadIdx.x - 1];

            if (abs(pixel.x - currentPixel.x) < threshold && abs(pixel.y - currentPixel.y) < threshold && abs(pixel.z - currentPixel.z) < threshold) {    
                if ((int) currentLabel.x + (int) currentLabel.y + (int) currentLabel.z < (int) label.x + (int) label.y + (int) label.z) {
                    hasUpdated[0] = 1; 
                    out[outIdx] = label;
                    return;
                }
            }
        }

        if (threadIdx.y+1 < 32) {
            uchar4 pixel = pixels[threadIdx.y + 1][threadIdx.x];
            uchar4 label = labels[threadIdx.y + 1][threadIdx.x];
            if (abs(pixel.x - currentPixel.x) < threshold && abs(pixel.y - currentPixel.y) < threshold && abs(pixel.z - currentPixel.z) < threshold) {    
                if ((int) currentLabel.x + (int) currentLabel.y + (int) currentLabel.z < (int) label.x + (int) label.y + (int) label.z) {
                    // labels[threadIdx.y][threadIdx.x] = label;
                    // __syncthreads();
                    hasUpdated[0] = 1; 
                    out[outIdx] = label;
                    return;
                }
            }
        }

        if (int (threadIdx.y)-1 > 0) {
            uchar4 pixel = pixels[threadIdx.y - 1][threadIdx.x];
            uchar4 label = labels[threadIdx.y - 1][threadIdx.x];
            if (abs(pixel.x - currentPixel.x) < threshold && abs(pixel.y - currentPixel.y) < threshold && abs(pixel.z - currentPixel.z) < threshold) {    
                if ((int) currentLabel.x + (int) currentLabel.y + (int) currentLabel.z < (int) label.x + (int) label.y + (int) label.z) {
                    hasUpdated[0] = 1; 
                    out[outIdx] = label;
                    return;
                }
            }
        }

        out[outIdx] = currentLabel; 
    }

    __global__ void labelComponentsMasterLabel(uchar4* input, uchar4* out, int width, int height, unsigned char* R, unsigned char* G, unsigned char* B, int threshold, unsigned char* hasUpdated) {
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x >= width) {
            return;
        }
        
        if (y >= height) {
            return;
        }

        unsigned char valR = R[y * width + x]; 
        unsigned char valG = G[y * width + x]; 
        unsigned char valB = B[y * width + x]; 

        uchar4 currentLabel = input[y * width + x];
    
        // find most right "master" label

        
        bool hasMaster = false;
        uchar4 masterLabel;
        for (int i = x + 1; i < width; i++) {
            unsigned char valRedRight = R[y * width + i];
            unsigned char valGreenRight = G[y * width + i];
            unsigned char valBlueRight = B[y * width + i];
            uchar4 labelRight = input[y * width + i];
            
            if (!(abs(valRedRight - valR) < threshold && abs(valGreenRight - valG) < threshold && abs(valBlueRight - valB) < threshold)) {            
                break;
            }

            if (((int) currentLabel.x + (int) currentLabel.y + (int) currentLabel.z < (int) labelRight.x + (int) labelRight.y + (int) labelRight.z)) {
                masterLabel = labelRight;
                hasMaster = true;
            }
        }

        if (hasMaster) {
            hasMaster = false;
            out[y * width + x] = masterLabel;
            hasUpdated[0] = 1;
            return;
        }

        for (int i = x - 1; i >= 0; i--) {
            unsigned char valRedLeft = R[y * width + i];
            unsigned char valGreenLeft = G[y * width + i];
            unsigned char valBlueLeft = B[y * width + i];
            uchar4 labelLeft = input[y * width + i];
            
            if (!(abs(valRedLeft - valR) < threshold && abs(valGreenLeft - valG) < threshold && abs(valBlueLeft - valB) < threshold)) {
                break;
            }
            
            if (((int) currentLabel.x + (int) currentLabel.y + (int) currentLabel.z < (int) labelLeft.x + (int) labelLeft.y + (int) labelLeft.z)) {
                masterLabel = labelLeft;
                hasMaster = true;
            }            
        }

        if (hasMaster) {
            hasMaster = false;
            out[y * width + x] = masterLabel;
            hasUpdated[0] = 1;
            return;
        }

        for (int i = y + 1; i < height; i++) {
            unsigned char valRedDown = R[i * width + x];
            unsigned char valGreenDown = G[i * width + x];
            unsigned char valBlueDown = B[i * width + x];
            uchar4 labelDown = input[i * width + x];
            
            if (!(abs(valRedDown - valR) < threshold && abs(valGreenDown - valG) < threshold && abs(valBlueDown - valB) < threshold)) {
                break;
            }

            if ((int) currentLabel.x + (int) currentLabel.y + (int) currentLabel.z < (int) labelDown.x + (int) labelDown.y + (int) labelDown.z) {
                masterLabel = labelDown;
                hasMaster = true;
            }
        }

        if (hasMaster) {
            hasMaster = false;
            out[y * width + x] = masterLabel;
            hasUpdated[0] = 1;
            return;
        }

        for (int i = y - 1; i >= 0; i--) {
            unsigned char valRedUp = R[i * width + x];
            unsigned char valGreenUp = G[i * width + x];
            unsigned char valBlueUp = B[i * width + x];
            uchar4 labelUp = input[i * width + x];
            
            if (!(abs(valRedUp - valR) < threshold && abs(valGreenUp - valG) < threshold && abs(valBlueUp - valB) < threshold)) {
                break;
            }

            if ((int) currentLabel.x + (int) currentLabel.y + (int) currentLabel.z < (int) labelUp.x + (int) labelUp.y + (int) labelUp.z) {
                masterLabel = labelUp;
                hasMaster = true;
            }
        }

        if (hasMaster) {
            hasMaster = false;
            out[y * width + x] = masterLabel;
            hasUpdated[0] = 1;
            return;
        }    

        out[y * width + x] = currentLabel;
    
    }
    
    __global__ void labelComponents(uchar4* input, uchar4* out, int width, int height, unsigned char *R, unsigned char *G, unsigned char *B, int threshold, unsigned char* hasUpdated) {
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x >= width) {
            return;
        }
        
        if (y >= height) {
            return;
        }
        

        // shared?
        unsigned char valR = R[y * width + x]; 
        unsigned char valG = G[y * width + x]; 
        unsigned char valB = B[y * width + x]; 

        uchar4 currentLabel = input[y * width + x];

        if (x+1 < width) {
            unsigned char valRedRight = R[y * width + x + 1];
            unsigned char valGreenRight = G[y * width + x + 1];
            unsigned char valBlueRight = B[y * width + x + 1];
            uchar4 labelRight = input[y * width + x + 1];
            
            if (abs(valRedRight - valR) < threshold && abs(valGreenRight - valG) < threshold && abs(valBlueRight - valB) < threshold) {
                
                if ((int) currentLabel.x + (int) currentLabel.y + (int) currentLabel.z < (int) labelRight.x + (int) labelRight.y + (int) labelRight.z) {
                    if (blockIdx.y == 0) {
                        // printf("label right: %d %d %d\n", labelRight.x, labelRight.y, labelRight.z);
                    }
                    // currentLabel = labelRight;

                    hasUpdated[0] = 1;
                    out[y * width + x] = labelRight;
                    //surf2Dwrite(labelRight, out, x * sizeof(char4), height -1 - y);
                    return;
                }
            }   
            //surf2Dwrite(currentLabel, out, x * sizeof(char4), height -1 - y);
        }

        if (x-1 < width) {
            unsigned char valRedRight = R[y * width + x - 1];
            unsigned char valGreenRight = G[y * width + x - 1];
            unsigned char valBlueRight = B[y * width + x - 1];
            uchar4 labelRight = input[y * width + x - 1];
            if (abs(valRedRight - valR) < threshold && abs(valGreenRight - valG) < threshold && abs(valBlueRight - valB) < threshold) {
                
                if ((int) currentLabel.x + (int) currentLabel.y + (int) currentLabel.z < (int) labelRight.x + (int) labelRight.y + (int) labelRight.z) {
                    // currentLabel = labelRight;
                    hasUpdated[0] = 1;
                    out[y * width + x] = labelRight;
                    return;
                }
            }   
        }

        if (y+1 < height) {
            unsigned char valRedBottom = R[(y+1) * width + x];
            unsigned char valGreenBottom = G[(y+1) * width + x];
            unsigned char valBlueBottom = B[(y+1) * width + x];
            uchar4 labelBottom = input[(y+1) * width + x];
            if (abs(valRedBottom - valR) < threshold && abs(valGreenBottom - valG) < threshold && abs(valBlueBottom - valB) < threshold) {
                if ((int) currentLabel.x + (int) currentLabel.y + (int) currentLabel.z < (int) labelBottom.x + (int) labelBottom.y + (int) labelBottom.z) {
                    // currentLabel = labelBottom;
                    hasUpdated[0] = 1;
                    out[y * width + x] = labelBottom;
                    return;
                }
            }
        }
        if (y-1 < height) {
            unsigned char valRedBottom = R[(y-1) * width + x];
            unsigned char valGreenBottom = G[(y-1) * width + x];
            unsigned char valBlueBottom = B[(y-1) * width + x];
            uchar4 labelBottom = input[(y-1) * width + x];
            if (abs(valRedBottom - valR) < threshold && abs(valGreenBottom - valG) < threshold && abs(valBlueBottom - valB) < threshold) {
                if ((int) currentLabel.x + (int) currentLabel.y + (int) currentLabel.z < (int) labelBottom.x + (int) labelBottom.y + (int) labelBottom.z) {
                    // currentLabel = labelBottom;
                    hasUpdated[0] = 1;
                    out[y * width + x] = labelBottom;
                    return;
                }
            }
        }

        out[y * width + x] = currentLabel;
    }

    __global__ void labelComponentsRowed(uchar4* input, uchar4* out, int width, int height, unsigned char *R, unsigned char *G, unsigned char *B) {
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x >= width) {
            return;
        }
        
        if (y >= height) {
            return;
        }

        // shared?
        unsigned char valR = R[y * width + x]; 
        unsigned char valG = G[y * width + x]; 
        unsigned char valB = B[y * width + x]; 

        int threshold = 30;
        uchar4 currentLabel = input[y * width + x];

        if (x+1 < width) {
            unsigned char valRedRight = R[y * width + x + 1];
            unsigned char valGreenRight = G[y * width + x + 1];
            unsigned char valBlueRight = B[y * width + x + 1];
            uchar4 labelRight = input[y * width + x + 1];
            
            if (abs(valRedRight - valR) < threshold && abs(valGreenRight - valG) < threshold && abs(valBlueRight - valB) < threshold) {
                if ((int) currentLabel.x + (int) currentLabel.y + (int) currentLabel.z < (int) labelRight.x + (int) labelRight.y + (int) labelRight.z) {
                    out[y * width + x] = labelRight;
                    return;
                }
            }   
        }

        if (y+1 < height) {
            unsigned char valRedBottom = R[(y+1) * width + x];
            unsigned char valGreenBottom = G[(y+1) * width + x];
            unsigned char valBlueBottom = B[(y+1) * width + x];
            uchar4 labelBottom = input[(y+1) * width + x];
            if (abs(valRedBottom - valR) < threshold && abs(valGreenBottom - valG) < threshold && abs(valBlueBottom - valB) < threshold) {
                if ((int) currentLabel.x + (int) currentLabel.y + (int) currentLabel.z < (int) labelBottom.x + (int) labelBottom.y + (int) labelBottom.z) {
                    out[y * width + x] = labelBottom;
                    return;
                }
            }
        }

        out[y * width + x] = currentLabel;
    }

    __global__ void copyToSurface(uchar4* input, hipSurfaceObject_t target, int width, int height) {
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x >= width) {
            return;
        }
        
        if (y >= height) {
            return;
        }
        
        uchar4 color = input[y * width + x];
        surf2Dwrite(color, target, x * sizeof(uchar4), height -1 - y);
    }

    __global__ void colorComponentAtPixel(hipTextureObject_t texture, hipSurfaceObject_t surface, int posX, int posY, int width, int height) {
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;

        __shared__ float4 toSearchLabel;
        if (threadIdx.x == 0) {
            toSearchLabel = tex2D<float4>(texture, posX, height -1 - posY);

        }
        __syncthreads();

        if (x >= width) {
            return;
        }
        
        if (y >= height) {
            return;
        }
        
        // uchar4 label = surf2Dread<uchar4>(surface, x * sizeof(uchar4), height -1 - y);
        float4 label = tex2D<float4>(texture, x, height -1 - y);

        float threshold = 0.019; // 0.019

        // mind that z is not used
        if (abs(toSearchLabel.x - label.x) < threshold && abs(toSearchLabel.y - label.y) < threshold && abs(toSearchLabel.z - label.z) < threshold) {
            uchar4 color = make_uchar4(0, 0, 255, 255);
            surf2Dwrite(color, surface, x * sizeof(uchar4), height -1 - y);
        }
        __syncthreads();

    }

    __global__ void colorComponentAtPixelExact(hipTextureObject_t texture, hipSurfaceObject_t surface, int posX, int posY, int width, int height, unsigned char R, unsigned char G, unsigned char B) {
        if (R == 0 && G == 0 && B == 255) {
            return;
        }
        
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x >= width) {
            return;
        }
        
        if (y >= height) {
            return;
        }

        uchar4 toSearchLabel = make_uchar4(R, G, B, 255);
        /*__shared__ uchar4 toSearchLabel;
        if (threadIdx.x == 0) {
            toSearchLabel = surf2Dread<uchar4>(surface, posX * sizeof(uchar4), height -1 - posY);
            // toSearchLabel = tex2D<uchar4>(texture, posX, posY);

        }
        __syncthreads();*/


        // uchar4 label = tex2D<uchar4>(texture, x, y);
        uchar4 label = surf2Dread<uchar4>(surface, x * sizeof(uchar4), height -1 - y);


        if (toSearchLabel.x == label.x && toSearchLabel.y == label.y && toSearchLabel.z == label.z) {
            uchar4 color = make_uchar4(0, 0, 255, 255);
            surf2Dwrite(color, surface, x * sizeof(uchar4), height -1 - y);
        }

    }

    __global__ void readPixelValue(uchar4* labels, int posX, int posY, unsigned char* R, unsigned char* G, unsigned char* B, int width, int height) {

        if (posX >= width) {
            return;
        }

        if (posY >= height) {
            return;
        }

        // uchar4 pixel = tex2D<uchar4>(texture, posX, posY);
        // uchar4 pixel = surf2Dread<uchar4>(surface, posX * sizeof(uchar4), height -1 -posY);
        uchar4 pixel = labels[posY * width + posX];
        R[0] = pixel.x;
        G[0] = pixel.y;
        B[0] = pixel.z;
    }
}